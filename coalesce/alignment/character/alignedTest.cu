#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "gputimer.cuh"
#include "cputimer.cuh"

#define NUMBER_OF_TESTS 10
#define DATA_SIZE 20L
#define ALIGNMENT_SIZE 4
#define VARIABLE_COUNT 4

typedef struct __align__(ALIGNMENT_SIZE) {
    #if VARIABLE_COUNT >= 1
    unsigned char a;
    #endif
    #if VARIABLE_COUNT >= 2
    unsigned char b;
    #endif
    #if VARIABLE_COUNT >= 3
    unsigned char c;
    #endif
    #if VARIABLE_COUNT >= 4
    unsigned char d;
    #endif
    #if VARIABLE_COUNT >= 5
    unsigned char e;
    #endif
    #if VARIABLE_COUNT >= 6
    unsigned char f;
    #endif
    #if VARIABLE_COUNT >= 7
    unsigned char g;
    #endif
    #if VARIABLE_COUNT >= 8
    unsigned char h;
    #endif
    #if VARIABLE_COUNT >= 9
    unsigned char i;
    #endif
    #if VARIABLE_COUNT >= 10
    unsigned char j;
    #endif
    #if VARIABLE_COUNT >= 11
    unsigned char k;
    #endif
    #if VARIABLE_COUNT >= 12
    unsigned char l;
    #endif
    #if VARIABLE_COUNT >= 13
    unsigned char m;
    #endif
    #if VARIABLE_COUNT >= 14
    unsigned char n;
    #endif
    #if VARIABLE_COUNT >= 15
    unsigned char o;
    #endif
    #if VARIABLE_COUNT >= 16
    unsigned char p;
    #endif
} AlignedData;

__global__ void alignedKernel (AlignedData* array, unsigned long long int numberOfElements) {
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int numberOfTotalThreads = gridDim.x * blockDim.x;
    unsigned long long int numberOfWorks = numberOfElements / numberOfTotalThreads;
    for (unsigned long long int i = 0; i < numberOfWorks; i += 1) {
        unsigned long long int index = idx * numberOfWorks + i;
        #if VARIABLE_COUNT >= 1
        array[index].a = array[index].a * 2;
        #endif
        #if VARIABLE_COUNT >= 2
        array[index].b = array[index].b * 2;
        #endif
        #if VARIABLE_COUNT >= 3
        array[index].c = array[index].c * 2;
        #endif
        #if VARIABLE_COUNT >= 4
        array[index].d = array[index].d * 2;
        #endif
        #if VARIABLE_COUNT >= 5
        array[index].e = array[index].e * 2;
        #endif
        #if VARIABLE_COUNT >= 6
        array[index].f = array[index].f * 2;
        #endif
        #if VARIABLE_COUNT >= 7
        array[index].g = array[index].g * 2;
        #endif
        #if VARIABLE_COUNT >= 8
        array[index].h = array[index].h * 2;
        #endif
        #if VARIABLE_COUNT >= 9
        array[index].i = array[index].i * 2;
        #endif
        #if VARIABLE_COUNT >= 10
        array[index].j = array[index].j * 2;
        #endif
        #if VARIABLE_COUNT >= 11
        array[index].k = array[index].k * 2;
        #endif
        #if VARIABLE_COUNT >= 12
        array[index].l = array[index].l * 2;
        #endif
        #if VARIABLE_COUNT >= 13
        array[index].m = array[index].m * 2;
        #endif
        #if VARIABLE_COUNT >= 14
        array[index].n = array[index].n * 2;
        #endif
        #if VARIABLE_COUNT >= 15
        array[index].o = array[index].o * 2;
        #endif
        #if VARIABLE_COUNT >= 16
        array[index].p = array[index].p * 2;
        #endif
    }
}

int main() {
    
    //Define Timers
    CPUTimer timer_aligned_malloc;
    CPUTimer timer_aligned_init;
    CPUTimer timer_aligned_free;


    GPUTimer timer_aligned_cudaMalloc;
    GPUTimer timer_aligned_HD;
    GPUTimer timer_aligned_kernel;
    GPUTimer timer_aligned_DH;
    GPUTimer timer_aligned_cudaFree;

    double aligned_malloc_sum = 0;
    double aligned_init_sum = 0;
    double aligned_free_sum = 0;

    float aligned_kernel_sum = 0;
    float aligned_cudaMalloc_sum = 0;
    float aligned_HD_sum = 0;
    float aligned_DH_sum = 0;
    float aligned_cudaFree_sum = 0;

    unsigned long long int number_of_elements = (DATA_SIZE * (1L << 30)) / sizeof(AlignedData);

    for (int j = 0; j < NUMBER_OF_TESTS; j++) {

        // Allocate host arrays

        timer_aligned_malloc.start();

        AlignedData* alignedHostArray = (AlignedData*) malloc (number_of_elements * sizeof(AlignedData));

        timer_aligned_malloc.end();
        aligned_malloc_sum += timer_aligned_malloc.elapsed();

        // Initialize arrays on the host

        timer_aligned_init.start();

        for (unsigned long long int i = 0; i < number_of_elements; i++) {
            #if VARIABLE_COUNT == 1
            alignedHostArray[i] = {1};
            #endif
            #if VARIABLE_COUNT == 2
            alignedHostArray[i] = {1, 2};
            #endif
            #if VARIABLE_COUNT == 3
            alignedHostArray[i] = {1, 2, 3};
            #endif
            #if VARIABLE_COUNT == 4
            alignedHostArray[i] = {1, 2, 3, 4};
            #endif
            #if VARIABLE_COUNT == 5
            alignedHostArray[i] = {1, 2, 3, 4, 5};
            #endif
            #if VARIABLE_COUNT == 6
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6};
            #endif
            #if VARIABLE_COUNT == 7
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7};
            #endif
            #if VARIABLE_COUNT == 8
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8};
            #endif
            #if VARIABLE_COUNT == 9
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
            #endif
            #if VARIABLE_COUNT == 10
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
            #endif
            #if VARIABLE_COUNT == 11
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11};
            #endif
            #if VARIABLE_COUNT == 12
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
            #endif
            #if VARIABLE_COUNT == 13
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13};
            #endif
            #if VARIABLE_COUNT == 14
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14};
            #endif
            #if VARIABLE_COUNT == 15
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
            #endif
            #if VARIABLE_COUNT == 16
            alignedHostArray[i] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
            #endif
        }

        timer_aligned_init.end();
        aligned_init_sum += timer_aligned_init.elapsed();

        // Allocate device memory
        AlignedData* d_alignedArray;

        timer_aligned_cudaMalloc.start();

        hipMalloc((void**)&d_alignedArray, number_of_elements * sizeof(AlignedData));

        timer_aligned_cudaMalloc.end();
        aligned_cudaMalloc_sum += timer_aligned_cudaMalloc.elapsed();

        // Copy host arrays to device

        timer_aligned_HD.start();

        hipMemcpy(d_alignedArray, alignedHostArray, number_of_elements * sizeof(AlignedData), hipMemcpyHostToDevice);

        timer_aligned_HD.end();
        aligned_HD_sum += timer_aligned_HD.elapsed();

        // Define block and grid sizes
        int blockSize = 256;
        int numBlocks = 196608 / blockSize;

        
        // Measure time for unaligned array processing

        timer_aligned_kernel.start();

        alignedKernel<<<numBlocks, blockSize>>>(d_alignedArray, number_of_elements);
        
        timer_aligned_kernel.end();
        aligned_kernel_sum += timer_aligned_kernel.elapsed();

        // Copy device arrays to host

        timer_aligned_DH.start();

        hipMemcpy(alignedHostArray, d_alignedArray, number_of_elements * sizeof(AlignedData), hipMemcpyDeviceToHost);

        timer_aligned_DH.end();
        aligned_DH_sum += timer_aligned_DH.elapsed();


        // Clean up

        timer_aligned_cudaFree.start();

        hipFree(d_alignedArray);

        timer_aligned_cudaFree.end();
        aligned_cudaFree_sum += timer_aligned_cudaFree.elapsed();

        timer_aligned_free.start();

        free(alignedHostArray);

        timer_aligned_free.end();
        aligned_free_sum += timer_aligned_free.elapsed();

    }

    printf("For %ldGB elements each %dbytes with %d alignment and %d tests {\n", DATA_SIZE, VARIABLE_COUNT, ALIGNMENT_SIZE, NUMBER_OF_TESTS);
    printf("Average malloc time aligned -> %lf\n", aligned_malloc_sum / NUMBER_OF_TESTS);
    printf("Average hipMalloc time aligned -> %lf\n", aligned_cudaMalloc_sum / NUMBER_OF_TESTS);
    printf("Average init time aligned -> %lf\n", aligned_init_sum / NUMBER_OF_TESTS);
    printf("Average HD time aligned -> %lf\n", aligned_HD_sum / NUMBER_OF_TESTS);
    printf("Average kernel time aligned -> %lf\n", aligned_kernel_sum / NUMBER_OF_TESTS);
    printf("Average DH time aligned -> %lf\n", aligned_DH_sum / NUMBER_OF_TESTS);
    printf("Average hipFree time aligned -> %lf\n", aligned_cudaFree_sum / NUMBER_OF_TESTS);
    printf("Average free time aligned -> %lf\n", aligned_free_sum / NUMBER_OF_TESTS);
    printf("}\n");

}