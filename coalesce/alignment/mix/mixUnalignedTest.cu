#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "gputimer.cuh"
#include "cputimer.cuh"

#define NUMBER_OF_TESTS 1
#define DATA_SIZE 20L

typedef struct {
    unsigned char a;
    unsigned char b;
    unsigned short int c;
    // unsigned int e;
    // unsigned int f;
    // unsigned int g;
    // unsigned int h;
} UnalignedData;


__global__ void unalignedKernel (UnalignedData* array, unsigned long long int numberOfElements) {
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int numberOfTotalThreads = gridDim.x * blockDim.x;
    unsigned long long int numberOfWorks = numberOfElements / numberOfTotalThreads;
    for (unsigned long long int i = 0; i < numberOfWorks; i += 1) {
        unsigned long long int index = idx * numberOfWorks + i;
        array[index].a = array[index].a * 2;
        array[index].b = array[index].b * 2;
        array[index].c = array[index].c * 2;
        // array[index].d = array[index].d * 2;
        // array[index].e = array[index].e * 2;
        // array[index].f = array[index].f * 2;
        // array[index].g = array[index].g * 2;
        // array[index].h = array[index].h * 2;
    }
}

int main() {
    
    //Define Timers
    CPUTimer timer_unaligned_malloc;
    CPUTimer timer_unaligned_init;
    CPUTimer timer_unaligned_free;


    GPUTimer timer_unaligned_cudaMalloc;
    GPUTimer timer_unaligned_HD;
    GPUTimer timer_unaligned_kernel;
    GPUTimer timer_unaligned_DH;
    GPUTimer timer_unaligned_cudaFree;

    double unaligned_malloc_sum = 0;
    double unaligned_init_sum = 0;
    double unaligned_free_sum = 0;

    float unaligned_kernel_sum = 0;
    float unaligned_cudaMalloc_sum = 0;
    float unaligned_HD_sum = 0;
    float unaligned_DH_sum = 0;
    float unaligned_cudaFree_sum = 0;

    unsigned long long int number_of_elements = (DATA_SIZE * (1L << 30)) / 4;

    for (int j = 0; j < NUMBER_OF_TESTS; j++) {
        // Allocate host arrays

        timer_unaligned_malloc.start();

        UnalignedData* unalignedHostArray = (UnalignedData*) malloc (number_of_elements * sizeof(UnalignedData));

        timer_unaligned_malloc.end();
        unaligned_malloc_sum += timer_unaligned_malloc.elapsed();

        // Initialize arrays on the host

        timer_unaligned_init.start();

        for (unsigned long long int i = 0; i < number_of_elements; i++) {
            unalignedHostArray[i] = {1, 2, 3};//, 4, 5, 6};//, 7, 8};
        }

        timer_unaligned_init.end();
        unaligned_init_sum += timer_unaligned_init.elapsed();

        // Allocate device memory

        UnalignedData* d_unalignedArray;

        timer_unaligned_cudaMalloc.start();

        hipMalloc((void**)&d_unalignedArray, number_of_elements * sizeof(UnalignedData));

        timer_unaligned_cudaMalloc.end();
        unaligned_cudaMalloc_sum += timer_unaligned_cudaMalloc.elapsed();

        // Copy host arrays to device

        timer_unaligned_HD.start();

        hipMemcpy(d_unalignedArray, unalignedHostArray, number_of_elements * sizeof(UnalignedData), hipMemcpyHostToDevice);

        timer_unaligned_HD.end();
        unaligned_HD_sum += timer_unaligned_HD.elapsed();

        // Define block and grid sizes

        int blockSize = 256;
        int numBlocks = 196608 / blockSize;

        
        // Measure time for unaligned array processing

        timer_unaligned_kernel.start();

        
        unalignedKernel<<<numBlocks, blockSize>>>(d_unalignedArray, number_of_elements);

        timer_unaligned_kernel.end();
        unaligned_kernel_sum += timer_unaligned_kernel.elapsed();

        // Copy device arrays to host

        timer_unaligned_DH.start();

        hipMemcpy(unalignedHostArray, d_unalignedArray, number_of_elements * sizeof(UnalignedData), hipMemcpyDeviceToHost);

        timer_unaligned_DH.end();
        unaligned_DH_sum += timer_unaligned_DH.elapsed();

        // Clean up

        timer_unaligned_cudaFree.start();

        hipFree(d_unalignedArray);

        timer_unaligned_cudaFree.end();
        unaligned_cudaFree_sum += timer_unaligned_cudaFree.elapsed();

        timer_unaligned_free.start();

        free(unalignedHostArray);

        timer_unaligned_free.end();
        unaligned_free_sum += timer_unaligned_free.elapsed();

    }

    printf("For %ldGB elements each %dbytes and %d tests {\n", DATA_SIZE, 12, NUMBER_OF_TESTS);
    printf("Average malloc time : unaligned -> %lf\n", unaligned_malloc_sum / NUMBER_OF_TESTS);
    printf("Average hipMalloc time : unaligned -> %lf\n", unaligned_cudaMalloc_sum / NUMBER_OF_TESTS);
    printf("Average init time : unaligned -> %lf\n", unaligned_init_sum / NUMBER_OF_TESTS);
    printf("Average HD time : unaligned -> %lf\n", unaligned_HD_sum / NUMBER_OF_TESTS);
    printf("Average kernel time : unaligned -> %lf\n", unaligned_kernel_sum / NUMBER_OF_TESTS);
    printf("Average DH time : unaligned -> %lf\n", unaligned_DH_sum / NUMBER_OF_TESTS);
    printf("Average hipFree time : unaligned -> %lf\n", unaligned_cudaFree_sum / NUMBER_OF_TESTS);
    printf("Average free time : unaligned -> %lf\n", unaligned_free_sum / NUMBER_OF_TESTS);
    printf("}\n");

}