#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t deviceProp;
    int device;

    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    printf("ConcurrentManagedAccess is %d\n", deviceProp.concurrentManagedAccess);

    return 0;
}
