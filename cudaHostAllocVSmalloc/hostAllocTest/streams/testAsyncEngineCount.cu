#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t deviceProp;
    int device;

    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    printf("AsyncEngineCount is %d\n", deviceProp.asyncEngineCount);

    return 0;
}
