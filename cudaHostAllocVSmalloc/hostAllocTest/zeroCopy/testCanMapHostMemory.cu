#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t deviceProp;
    int device;

    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    printf("canMapHostMemory is %d\n", deviceProp.canMapHostMemory);

    return 0;
}
