#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "gputimer.cuh"
#include "cputimer.cuh"

#define NUMBER_OF_TESTS 10
#define DATA_SIZE 8L
#define ITERATIONS 10000


__global__ void kernel(int* input, int* output, unsigned long long int numberOfElements) {
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int numberOfTotalThreads = (unsigned long long int) gridDim.x * blockDim.x;
    unsigned long long int numberOfWorks = numberOfElements / numberOfTotalThreads;
    for (unsigned long long int i = 0; i < numberOfWorks; ++i) {
        unsigned long long int index = idx * numberOfWorks + i;
        int cond = (index % 10 != 0);
        int dest = 0;
        int src = input[index];
        int j;
        if (cond) {
            for (j = 0; j < ITERATIONS; j++) {
                dest += src * src + 5 * src + 6;
                dest *= 3;
                dest += src * 2;
            }
        }
        else {
            for (j = 0; j < ITERATIONS; j++) {
                dest += src * src + src * 2 + 1;
                dest *= 3;
                dest += src * 3;
            }
        }
        output[index] = dest;
    }
}

int main () {
    
    GPUTimer timer_cudaMalloc;
    GPUTimer timer_HD;
    GPUTimer timer_kernel;
    GPUTimer timer_DH;
    GPUTimer timer_cudaFree;
    GPUTimer timer_total;

    CPUTimer timer_malloc;
    CPUTimer timer_init;
    CPUTimer timer_free;


    float cudaMalloc_sum = 0;
    float HD_sum = 0;
    float kernel_sum = 0;
    float DH_sum = 0;
    float cudaFree_sum = 0;
    float total_sum = 0;

    double malloc_sum = 0;
    double init_sum = 0;
    double free_sum = 0;

    for(int j = 0; j < NUMBER_OF_TESTS; j++) {        

        timer_total.start();

        int *h_input, *h_output;
        unsigned long long int numberOfElements = ((DATA_SIZE / 2) * (1L << 30)) / sizeof(int);

        timer_malloc.start();


        h_input = (int *) malloc(numberOfElements * sizeof(int));
        h_output = (int *) malloc(numberOfElements * sizeof(int));

        timer_malloc.end();
        malloc_sum += timer_malloc.elapsed();
    
        timer_init.start();

        for (unsigned long long int i = 0; i < numberOfElements; i++) {
            h_input[i] = (int) (i % (1 << 29));
        }

        timer_init.end();
        init_sum += timer_init.elapsed();

        int *d_input, *d_output;

        timer_cudaMalloc.start();

        hipMalloc((void **) &d_input, numberOfElements * sizeof(int));
        hipMalloc((void **) &d_output, numberOfElements * sizeof(int));

        timer_cudaMalloc.end();
        cudaMalloc_sum += timer_cudaMalloc.elapsed();

        timer_HD.start();

        hipMemcpy((void *) d_input, (void *) h_input, numberOfElements * sizeof(int), hipMemcpyHostToDevice);

        timer_HD.end();
        HD_sum += timer_HD.elapsed();

        timer_kernel.start();

        int blockSize = 256;
        int numBlocks = 196608 / blockSize;
        kernel<<<numBlocks, blockSize>>>(d_input, d_output, numberOfElements);
        hipDeviceSynchronize();

        timer_kernel.end();
        kernel_sum += timer_kernel.elapsed();

        timer_DH.start();

        hipMemcpy((void *) h_output, (void *) d_output, numberOfElements * sizeof(int), hipMemcpyDeviceToHost);

        timer_DH.end();
        DH_sum += timer_DH.elapsed();

        timer_free.start();

        free(h_input);
        free(h_output);

        timer_free.end();
        free_sum += timer_free.elapsed();

        timer_cudaFree.start();

        hipFree(d_input);
        hipFree(d_output);

        timer_cudaFree.end();
        cudaFree_sum += timer_cudaFree.elapsed();

        timer_total.end();
        total_sum += timer_total.elapsed();

    }

    printf("Average malloc time for size %ldGB is %f\n", DATA_SIZE, malloc_sum / NUMBER_OF_TESTS);
    printf("Average init time for size %ldGB is %f\n", DATA_SIZE, init_sum / NUMBER_OF_TESTS);
    printf("Average hipMalloc time for size %ldGB is %f\n", DATA_SIZE, cudaMalloc_sum / NUMBER_OF_TESTS);
    printf("Average HD time for size %ldGB is %f\n", DATA_SIZE, HD_sum / NUMBER_OF_TESTS);
    printf("Average kernel time for size %ldGB is %f\n", DATA_SIZE, kernel_sum / NUMBER_OF_TESTS);
    printf("Average DH time for size %ldGB is %f\n", DATA_SIZE, DH_sum / NUMBER_OF_TESTS);
    printf("Average free time for size %ldGB is %f\n", DATA_SIZE, free_sum / NUMBER_OF_TESTS);
    printf("Average hipFree time for size %ldGB is %f\n", DATA_SIZE, cudaFree_sum / NUMBER_OF_TESTS);
    printf("Average total time for size %ldGB is %f\n", DATA_SIZE, total_sum / NUMBER_OF_TESTS);

    return 0;
}