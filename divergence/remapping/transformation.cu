#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "gputimer.cuh"
#include "cputimer.cuh"

#define NUMBER_OF_TESTS 10
#define DATA_SIZE 8L
#define ITERATIONS 10000


__global__ void kernel(int* input, int* output, unsigned long long int numberOfElements) {
    unsigned long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int numberOfTotalThreads = (unsigned long long int) gridDim.x * blockDim.x;
    unsigned long long int numberOfWorks = numberOfElements / numberOfTotalThreads;
    for (unsigned long long int i = 0; i < numberOfWorks; ++i) {
        unsigned long long int index = idx * numberOfWorks + i;
        int src = input[index];
        int cond = (src % 10 != 0);
        int dest = 0;
        int j;
        if (cond) {
            for (j = 0; j < ITERATIONS; j++) {
                dest += src * src + 5 * src + 6;
                dest *= 3;
                dest += src * 2;
            }
        }
        else {
            for (j = 0; j < ITERATIONS; j++) {
                dest += src * src + src * 2 + 1;
                dest *= 3;
                dest += src * 3;
            }
        }
        output[index] = dest;
    }
}

void transform(int* input, int* transformed_input, unsigned long long int numberOfElements) {
    unsigned long long int start_index = 0;
    unsigned long long int end_index = numberOfElements - 1;
    for (unsigned long long int i = 0; i < numberOfElements; i++) {
        int value = input[i];
        if (value % 10 != 0) {
            transformed_input[start_index] = value;
            start_index += 1;
        } else {
            transformed_input[end_index] = value;
            end_index -= 1;
        }
    }
}

int main () {
    GPUTimer timer_kernel;
    GPUTimer timer_total;
    
    CPUTimer timer_transform;

    float kernel_sum = 0;
    float total_sum = 0;
    
    double transform_sum = 0;

    for(int j = 0; j < NUMBER_OF_TESTS; j++) {        

        timer_total.start();

        int *input;
        int *h_input, *h_output;
        unsigned long long int numberOfElements = ((DATA_SIZE / 2) * (1L << 30)) / sizeof(int);

        input = (int *) malloc(numberOfElements * sizeof(int));
        h_input = (int *) malloc(numberOfElements * sizeof(int));
        h_output = (int *) malloc(numberOfElements * sizeof(int));
    

        for (unsigned long long int i = 0; i < numberOfElements; i++) {
            input[i] = (int) (i % (1 << 29));
        }

        timer_transform.start();

        transform(input, h_input, numberOfElements);

        timer_transform.end();
        transform_sum += timer_transform.elapsed();

        int *d_input, *d_output;

        hipMalloc((void **) &d_input, numberOfElements * sizeof(int));
        hipMalloc((void **) &d_output, numberOfElements * sizeof(int));

        hipMemcpy((void *) d_input, (void *) h_input, numberOfElements * sizeof(int), hipMemcpyHostToDevice);

        timer_kernel.start();

        int blockSize = 256;
        int numBlocks = 196608 / blockSize;
        kernel<<<numBlocks, blockSize>>>(d_input, d_output, numberOfElements);
        hipDeviceSynchronize();

        timer_kernel.end();
        kernel_sum += timer_kernel.elapsed();


        hipMemcpy((void *) h_output, (void *) d_output, numberOfElements * sizeof(int), hipMemcpyDeviceToHost);

        free(h_input);
        free(h_output);

        hipFree(d_input);
        hipFree(d_output);

        timer_total.end();
        total_sum += timer_total.elapsed();

    }

    printf("Average transform time for size %ldGB is %f\n", DATA_SIZE, transform_sum / NUMBER_OF_TESTS);
    printf("Average kernel time for size %ldGB is %f\n", DATA_SIZE, kernel_sum / NUMBER_OF_TESTS);
    printf("Average total time for size %ldGB is %f\n", DATA_SIZE, total_sum / NUMBER_OF_TESTS);

    return 0;
}